#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <ctime>
#include <vector>

#include <SDL.h>

#include "bitmap.hh"
#include "gui.hh"
#include "star.hh"
#include "util.hh"
#include "vec2d.hh"

__global__ void addx(vec2d* newX) {
  *newX += vec2d(1, 0);
}

int main() {

  vec2d center = vec2d(0, 0);
  vec2d* newX;
  
  if(hipMalloc(&newX, sizeof(vec2d)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate newcenter on GPU\n");
    exit(2);
  }

  if(hipMemcpy(newX, &center, sizeof(vec2d), hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy center to the GPU\n");
  }

  addx<<<3, 1>>>(newX);

  hipDeviceSynchronize();

  if(hipMemcpy(&center, newX, sizeof(vec2d), hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy newcenter from the GPU\n");
  }

  printf("%f\n", center.x());

  hipFree(newX);

  return 0;
}
